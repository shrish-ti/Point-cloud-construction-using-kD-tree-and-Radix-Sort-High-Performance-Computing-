#include "hip/hip_runtime.h"
%%cu
#include <iostream>
#define loopy cout<<"loopy"<<" "
#define pp pair<int,int>
#define pb push_back
#define sp " "
#define ll long long int
#define nl "\n"
#define pf cout<<
#define fir first
#define sec second
#include<cstdio>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono>
using namespace std::chrono;
using namespace std;
#include <chrono>
using namespace std::chrono;
#define BLOCK 3
#define THREAD 1024
#define MAX_DIM 3
struct Node {
    double val[3];
    struct Node* left, * right;
    Node()
    {
        for (int i = 0; i < MAX_DIM; i++)
        {
            val[i] = 0;
        }
        left = NULL;
        right = NULL;
    }
};
void insertionSort(int h_array[][MAX_DIM], int numberofElements, int dim, int idx)  
{  
    int i, key, temp1, temp2, j;  
    
    for (i = 1; i<=numberofElements-1; i++) 
    {  
        
        key = h_array[i][idx];
        if(idx==0)
        {
        temp1=h_array[i][1];
        temp2=h_array[i][2];
        }
        else if(idx==1)
        {
        temp1=h_array[i][0];
        temp2=h_array[i][2];
        }
        else
        {
            temp1=h_array[i][0];
            temp2=h_array[i][1];
        }
        j = i - 1; 
        while (j >= 0 && h_array[j][idx] > key) 
        {  
            h_array[j + 1][idx]=h_array[j][idx];
            if(idx==0)
            {
            h_array[j+1][1]=h_array[j][1];
            h_array[j+1][2]=h_array[j][2];
            }
            else if(idx==1)
            {
            h_array[j+1][0]=h_array[j][0];
            h_array[j+1][2]=h_array[j][2];
            }
            else
            {
                h_array[j+1][0]=h_array[j][0];
                h_array[j+1][1]=h_array[j][1];
            }
            j = j - 1;  
        }  
        h_array[j + 1][idx] = key;  
        
        if(idx==0)
        {
        h_array[j+1][1]=temp1;
        h_array[j+1][2]=temp2;
        }
        
        else if(idx==1)
        {
        h_array[j+1][0]=temp1;
        h_array[j+1][2]=temp2;
        }
        
        else
        {
        h_array[j+1][0]=temp1;
        h_array[j+1][1]=temp2;
        }
    }
}

int getMax(int h_array[][MAX_DIM], int numberOfElements, int dim, int idx) 
{   
	int mx = h_array[0][idx]; 
	for (int i = 0; i < numberOfElements; i++) 
		if (h_array[i][idx] > mx) 
			mx = h_array[i][idx]; 
	return mx; 
	
} 

void countSort(int h_array[][MAX_DIM], int numberOfElements, int dim, int idx, int exp) 
{ 
	int output[numberOfElements][dim]; 
	int i, count[10] = { 0 }; 

	for (i = 0; i < numberOfElements; i++) 
		count[(h_array[i][idx] / exp) % 10]++; 

	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	for (i = numberOfElements - 1; i >= 0; i--) { 
		output[count[(h_array[i][idx] / exp) % 10] - 1][idx] = h_array[i][idx];
		if(idx==0)
		{
		output[count[(h_array[i][idx] / exp) % 10] - 1][1] = h_array[i][1];
		output[count[(h_array[i][idx] / exp) % 10] - 1][2] = h_array[i][2];
		}
		else if(idx==1)
		{
		output[count[(h_array[i][idx] / exp) % 10] - 1][0] = h_array[i][0];
		output[count[(h_array[i][idx] / exp) % 10] - 1][2] = h_array[i][2];
		}
		else
		{
		    output[count[(h_array[i][idx] / exp) % 10] - 1][0] = h_array[i][0];
		    output[count[(h_array[i][idx] / exp) % 10] - 1][1] = h_array[i][1];
		}
		count[(h_array[i][idx] / exp) % 10]--; 
	} 

	for (i = 0; i < numberOfElements; i++) 
	{
		h_array[i][0] = output[i][0]; 
		h_array[i][1]=output[i][1];
		h_array[i][2]=output[i][2];
}
    
} 

void sradixsort(int h_array[][MAX_DIM], int numberOfElements, int dim, int idx) 
{ 
	int m = getMax(h_array, numberOfElements, dim, idx); 

	for (int exp = 1; m / exp > 0; exp *= 10) 
		countSort(h_array, numberOfElements, dim, idx, exp); 
} 

__device__ int function(int value, int bit, int bitset)
{
    if (bitset == 1)
    {
        if ((value & bit) != 0)
        {
            return 1;
        }
        else
            return 0;
    }
    else
    {
        if ((value & bit) == 0)
        {
            return 1;
        }
        else
        {
            return 0;
        }
    }
}
__global__ void predicateDevice(int* d_array, int* d_predicateArrry, int d_numberOfElements, int bit, int bitset, int dim, int idx)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_numberOfElements)
    {

        d_predicateArrry[index] = function(d_array[dim * index + idx], bit, bitset);
    }
}

__global__ void scatter(int* d_array, int* d_scanArray, int* d_predicateArrry, int* d_scatteredArray, int d_numberOfElements, int offset, int dim, int idx)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_numberOfElements)
    {
        if (d_predicateArrry[index] == 1)
        {
            for (int i = 0; i < dim; i++)
            {
                d_scatteredArray[dim * (d_scanArray[index] - 1 + offset) + i] = d_array[dim * index + i];
            }
            /*
            d_scatteredArray[dim*(d_scanArray[index] - 1 + offset)] = d_array[dim*index];
            d_scatteredArray[dim*(d_scanArray[index] - 1 + offset) +1] = d_array[dim*index +1];
            */
        }
    }
}
__global__ void hillisSteeleScanDevice(int* d_array, int numberOfElements, int* d_tmpArray, int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index > numberOfElements)
    {
        return;
    }
    d_tmpArray[index] = d_array[index];
    if (index - moveIndex >= 0)
    {

        d_tmpArray[index] = d_tmpArray[index] + d_array[index - moveIndex];
    }
}
int* hillisSteeleScanHost(int* d_scanArray, int numberOfElements)
{


    int* d_tmpArray;
    int* d_tmpArray1;
    hipMalloc(&d_tmpArray1, sizeof(int) * numberOfElements);
    hipMalloc(&d_tmpArray, sizeof(int) * numberOfElements);
    hipMemcpy(d_tmpArray1, d_scanArray, sizeof(int) * numberOfElements, hipMemcpyDeviceToDevice);
    int j, k = 0;
    for (j = 1; j < numberOfElements; j = j * 2, k++)
    {
        if (k % 2 == 0)
        {
            hillisSteeleScanDevice << <BLOCK, THREAD >> > (d_tmpArray1, numberOfElements, d_tmpArray, j);
            hipDeviceSynchronize();
        }
        else
        {
            hillisSteeleScanDevice << <BLOCK, THREAD >> > (d_tmpArray, numberOfElements, d_tmpArray1, j);
            hipDeviceSynchronize();
        }
    }
    hipDeviceSynchronize();
    if (k % 2 == 0)
    {

        return d_tmpArray1;
    }
    else
    {
        return d_tmpArray;
    }
}
__global__ void print(int* d_predicateArrry, int numberOfElements)
{

    for (int i = 0; i < numberOfElements; i++)
    {
        printf("index = %d value = %d\n", i, d_predicateArrry[i]);
    }
}

int* compact(int* d_array, int numberOfElements, int bit, int dim, int idx)
{
    int offset;
    int* d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry, sizeof(int) * numberOfElements);
    predicateDevice << <BLOCK, THREAD >> > (d_array, d_predicateArrry, numberOfElements, bit, 0, dim, idx);
    int* d_scanArray;
    d_scanArray = hillisSteeleScanHost(d_predicateArrry, numberOfElements);
    int* d_scatteredArray;
    hipMalloc((void**)&d_scatteredArray, sizeof(int) * numberOfElements * dim);
    //cout<<"offset = "<<offset<<"\n";
    scatter << <BLOCK, THREAD >> > (d_array, d_scanArray, d_predicateArrry, d_scatteredArray, numberOfElements, 0, dim, idx);
    hipMemcpy(&offset, d_scanArray + numberOfElements - 1, sizeof(int), hipMemcpyDeviceToHost);
    predicateDevice << <BLOCK, THREAD >> > (d_array, d_predicateArrry, numberOfElements, bit, 1, dim, idx);
    d_scanArray = hillisSteeleScanHost(d_predicateArrry, numberOfElements);
    scatter << <BLOCK, THREAD >> > (d_array, d_scanArray, d_predicateArrry, d_scatteredArray, numberOfElements, offset, dim, idx);
    return d_scatteredArray;
}
int offset;
int* positivenegativesplit(int* d_array, int numberOfElements, int bit, int bitset, int dim, int idx)
{   
    /*
    int blockSize;      
    int minGridSize;    
    int gridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, predicateDevice, 0, numberOfElements);
    gridSize = (numberOfElements + blockSize - 1) / blockSize;
    cout << gridSize << sp << blockSize << nl;
    */
    int* d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry, sizeof(int) * numberOfElements);
    predicateDevice << <BLOCK, THREAD >> > (d_array, d_predicateArrry, numberOfElements, bit, bitset, dim, idx);
    /*
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    */
    int* d_scanArray;
    d_scanArray = hillisSteeleScanHost(d_predicateArrry, numberOfElements);
    int* d_scatteredArray;
    hipMemcpy(&offset, d_scanArray + numberOfElements - 1, sizeof(int), hipMemcpyDeviceToHost);
    //cout << offset << "\n";
    hipMalloc((void**)&d_scatteredArray, sizeof(int) * offset * dim);
    scatter << <BLOCK, THREAD >> > (d_array, d_scanArray, d_predicateArrry, d_scatteredArray, numberOfElements, 0, dim, idx);
    return d_scatteredArray;
}
int* radixSort(int* d_array, int numberOfElements, int dim, int idx)
{
    int bit;
    int* d_negativeArray = positivenegativesplit(d_array, numberOfElements, 1L << 31, 1, dim, idx);
    for (int i = 0; i < sizeof(int) * 8; i++)
    {
        bit = 1 << i;
        d_negativeArray = compact(d_negativeArray, offset, bit, dim, idx);
    }
    int* d_postiveArray = positivenegativesplit(d_array, numberOfElements, 1L << 31, 0, dim, idx);

    /*
    int* temp = new int[6];
    hipMemcpy(temp, d_postiveArray, sizeof(int) * 3 * 2, hipMemcpyDeviceToHost);
    for (int i = 0; i < 6; i++)cout << temp[i] << " ";
    cout << "\n";
    */

    for (int i = 0; i < sizeof(int) * 8; i++)
    {
        bit = 1 << i;
        d_postiveArray = compact(d_postiveArray, offset, bit, dim, idx);
        /*
        hipMemcpy(temp, d_postiveArray, sizeof(int) * 3 * 2, hipMemcpyDeviceToHost);
        for (int i = 0; i < 6; i++)cout << temp[i] << " ";
        cout << "\n";
        */
    }

    hipMemcpy(d_array, d_negativeArray, sizeof(int) * (numberOfElements - offset) * dim, hipMemcpyDeviceToDevice);
    hipMemcpy(d_array + (numberOfElements - offset), d_postiveArray, sizeof(int) * offset * dim, hipMemcpyDeviceToDevice);
    return d_array;
}
void sort(int h_array[][3], int numberOfElements, int dim, int idx)
{
    int* d_array;
    hipMalloc((void**)&d_array, sizeof(int) * numberOfElements * dim);
    hipMemcpy(d_array, h_array, sizeof(int) * numberOfElements * dim, hipMemcpyHostToDevice);
   
    d_array = radixSort(d_array, numberOfElements, dim, idx);
   
    hipMemcpy(h_array, d_array, sizeof(int) * numberOfElements * dim, hipMemcpyDeviceToHost);

}

Node* make_tree( int h_array[][3], int numberOfElements, int dim, int idx)
{
    //cout << start << sp << end << nl;
    if (numberOfElements<=0)return NULL;
    if (numberOfElements == 1)
    {
        Node* root = new Node();
        root->val[0] = *(*(h_array + 0) + 0);
        root->val[1] = *(*(h_array + 0) + 1);
        root->val[2] = *(*(h_array + 0) + 2);
        //cout << root->val[0] << sp << root->val[1] << nl;
        return root;
    }

    if(numberOfElements>=1024)
 {
    sort(h_array, numberOfElements, dim, idx);
 }
 else if(16<=numberOfElements&&numberOfElements<1024) 
 {
     sradixsort(h_array, numberOfElements, dim, idx);
 } 
 else
 {
     insertionSort(h_array, numberOfElements, dim, idx);
 }
    
    int md = numberOfElements / 2;
    int count2 = md - 1;
    if (numberOfElements % 2 == 1)count2 = md;
    
    Node* root = new Node();
    
    root->val[0] = *(*(h_array + md) + 0);
    root->val[1] = *(*(h_array + md) + 1);
    root->val[2] = *(*(h_array + md) + 2);
    //cout << root->val[0] << sp << root->val[1] << nl;

    root->left = make_tree(h_array, md, dim, (idx + 1) % dim);
    root->right = make_tree(h_array+md+1, count2, dim, (idx + 1) % dim);
    

    return root;

}

int main()
{
    int numberOfElements = 200000;
    int dim = MAX_DIM;
    cout << "The dimensions of mtrix are " << numberOfElements << " x " << dim << " \n";
    int h_array[numberOfElements][MAX_DIM];
    //cout << "Enter the elemets of the matrix \n";
    for (int i = 0; i < numberOfElements; i++)
    {
        h_array[i][0] = rand()%5000;
        h_array[i][1] = rand()%5000; 
        h_array[i][2] = rand()%5000;
    }
    auto start = high_resolution_clock::now();
    cout << "Cons start!!!";
    Node* root = make_tree(h_array, numberOfElements, dim, 0);
    cout << "Cons end!!!";
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << nl;
    cout << "Test duration is " << duration.count()/1000 << endl;
     

}
