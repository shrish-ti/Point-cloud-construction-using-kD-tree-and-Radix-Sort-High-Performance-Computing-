#include <bits/stdc++.h>
using namespace std;
int main()
{
//Basic declaration
int NoOfElements=2000;
int *h_array= new int[NoOfElements];
int *d_array;
for(int i=0; i<NoOfElements; i++)
{
h_array[i]=rand()%1000;
}

//Allocation on GPU
hipMalloc((void**)&d_array, sizeof(int)*NoOfElements);

//Copying from Device to GPU
hipMemcpy(d_array, h_array,sizeof(int)*NoOfElements, hipMemcpyHostToDevice);

//Calling kernel
d_array=radixSort(d_array, NoOfElements);

//Copying from GPU to CPU
hipMemcpy(h_array, d_array,sizeof(int)*NoOfElements,hipMemcpyDeviceToHost);

}
